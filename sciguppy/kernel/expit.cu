
#include <hip/hip_runtime.h>
__global__ void expit_kernel(float *d_a, float *d_aout, int size) {
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= size) {
        return;
    }

    const float x = d_a[id];
    float tmp;
    if (x < 0) {
        tmp = expf(x);
        d_aout[id] = tmp / (1.0 + tmp);
    } else {
        d_aout[id] = 1.0 / (1.0 + expf(-x));
    }
}

__global__ void expit_fast_kernel(float *d_a, float *d_aout, int size) {
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= size) {
        return;
    }

    const float x = d_a[id];
    float tmp;
    if (x <= -6) {
        d_aout[id] = 0;
    } else if (x >= 6) {
        d_aout[id] = 1;
    } else if (x < 0) {
        tmp = __expf(x);
        d_aout[id] = tmp / (1.0 + tmp);
    } else {
        d_aout[id] = 1.0 / (1.0 + __expf(-x));
    }
}

__global__ void expit_back_kernel(float *d_a, float *d_err, float *d_out, int size) {
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= size) {
        return;
    }

    const float x = d_a[id];
    d_out[id] = x * (1-x) * d_err[id];
}

__global__ void exp_fast_kernel(float *d_a, float *d_aout, int size) {
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= size) {
        return;
    }

    const float x = d_a[id];
    if (x <= -87) {
        d_aout[id] = 1.6458115E-38;
    } else if (x >= 87) {
        d_aout[id] = 6.0760303E+37;
    } else {
        d_aout[id] = __expf(x);
    }
}
